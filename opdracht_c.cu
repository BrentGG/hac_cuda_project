#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <curl/curl.h>
#include <string.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define POOLSTRIDE 2

struct Pixel
{
    unsigned char r, g, b, a;
};

void convoluteCPU(unsigned char* input, unsigned char* output, int width, int height, float kernel[3][3])
{
    for (int row = 0; row < height - 2; row++) {
        for (int col = 0; col < width - 2; col++) {
            int sum[4] = {0, 0, 0, 0};
            int opacity = 0;
            for (int i = 0; i < 3; ++i) {
                for (int j = 0; j < 3; ++j) {
                    Pixel* p = (Pixel*)&input[((row + 1) + (i - 1)) * width * 4 + 4 * ((col + 1) + (j - 1))];
                    sum[0] += p->r * kernel[i][j];
                    sum[1] += p->g * kernel[i][j];
                    sum[2] += p->b * kernel[i][j];
                    sum[3] += p->a * kernel[i][j];
                    if (i == 1 || j == 1)
                        opacity = p->a;
                }
            }
            Pixel* ptrPixel = (Pixel*)&output[row * width * 4 + 4 * col];
            for (int i = 0; i < 3; ++i) {
                if (sum[i] < 0)
                    sum[i] = 0;
                else if (sum[i] > 255)
                    sum[i] = 255;
            }
            ptrPixel->r = sum[0];
            ptrPixel->g = sum[1];
            ptrPixel->b = sum[2];
            ptrPixel->a = opacity;
        }
    }
}

void poolCPU(unsigned char* input, unsigned char* outputMaxPool, unsigned char* outputMinPool, unsigned char* outputAvgPool, int width, int height, int poolStride) {
    int poolWidth = (int)(width / poolStride);
    int poolHeight = (int)(height / poolStride);
    int row = 0;
    for (int i = 0; i < height; i += poolStride) {
        int col = 0;
        for (int j = 0; j < width; j += poolStride) {
            Pixel* p = (Pixel*)&input[i * width * 4 + 4 * j];
            int max[3] = {p->r, p->g, p->b};
            int min[3] = {p->r, p->g, p->b};
            float avg[3] = {0, 0, 0};
            for (int k = i; k < i + poolStride; ++k) {
                for (int l = j; l < j + poolStride; ++l) {
                    Pixel* q = (Pixel*)&input[k * width * 4 + 4 * l];
                    int values[3] = {q->r, q->g, q->b};
                    for (int v = 0; v < 3; ++v) {
                        if (values[v] > max[v])
                            max[v] = values[v];
                        if (values[v] < min[v])
                            min[v] = values[v];
                        avg[v] += values[v];
                    }
                }
            }
            for (int v = 0; v < 3; ++v)
                avg[v] /= poolStride * poolStride;
            Pixel* q = (Pixel*)&outputMaxPool[row * poolWidth * 4 + 4 * col];
            q->r = max[0];
            q->g = max[1];
            q->b = max[2];
            q->a = 255;
            Pixel* r = (Pixel*)&outputMinPool[row * poolWidth * 4 + 4 * col];
            r->r = min[0];
            r->g = min[1];
            r->b = min[2];
            r->a = 255;
            Pixel* s = (Pixel*)&outputAvgPool[row * poolWidth * 4 + 4 * col];
            s->r = round(avg[0]);
            s->g = round(avg[1]);
            s->b = round(avg[2]);
            s->a = 255;
            ++col;
        }
        ++row;
    }
}

static size_t write_data(void *ptr, size_t size, size_t nmemb, void *stream)
{
  size_t written = fwrite(ptr, size, nmemb, (FILE *)stream);
  return written;
}

void printTime(float seconds) {
    int m = (int)(seconds / 60);
    int s = (int)(seconds - (m * 60));
    int ms = (int)((seconds - (m * 60) - s) * 1000);
    if (m > 0)
        printf("%dm ", m);
    if (s > 0 || m > 0)
        printf("%ds ", s);
    if (ms > 0)
        printf("%dms", ms);
    else if (m == 0 && s == 0)
        printf("%fms", seconds * 1000);
}

int main(int argc, char** argv)
{
    clock_t programStart = clock();

    // Image URLs and names
    int imageAmount = 10;
    char urls[imageAmount][1024] = {
        "https://images.nintendolife.com/835d44208f0a6/mario-movie.large.jpg", // mario
        "https://img.buzzfeed.com/buzzfeed-static/static/2022-03/16/0/enhanced/c1de3db394fb/original-1460-1647389279-4.png", // the rock
        "https://images.squarespace-cdn.com/content/v1/5b788d28697a98e17a6d4c7a/b83f0eab-7dd6-4e9b-83a1-13139ac2a03b/rickroll+cropped.png", // rick astley
        "https://a-z-animals.com/media/2021/02/Kinkajou-header.jpg", //kinkajou
        "https://static.wikia.nocookie.net/marveldatabase/images/6/64/Incredible_Hulk_Vol_2_75_Textless.jpg/revision/latest/scale-to-width-down/300?cb=20050830175533", //hulk
        "https://laughingsquid.com/wp-content/uploads/bert-20110421-082506.jpg", //bert
        "https://car-anwb.akamaized.net/aas-afbeeldingen/117222g.jpg?imwidth=760&imheight=500", //ford
        "https://i.pinimg.com/originals/41/0b/2e/410b2eb9e59520a7ad4de7aa4fc9f722.jpg", //micky minnie mouse
        "https://pbs.twimg.com/media/CDN2AnCWYAIquzo.jpg", //canadian poop
        "https://media.istockphoto.com/id/1153678999/nl/vector/de-titel-van-het-eind-handschrift-op-rode-ronde-bacground-oude-film-einde-scherm-vector.jpg?s=612x612&w=0&k=20&c=OmFVqkf5TgXxizS0pWq5lEgbNQTNSny5W-BPmBahr1I=", //the end
        //extra URL links 
        /*
        "https://images0.persgroep.net/rcs/MP9RjFYsAOAE1ArHRLyorkHlKXU/diocontent/203486167/_fitwidth/1240?appId=93a17a8fd81db0de025c8abd1cca1279&quality=0.9",  //biden
        "https://cdn.shopify.com/s/files/1/0351/9630/5545/products/Good-Smile-Company-Pocket-Maquette-Demon-Slayer-Kimetsu-no-Yaiba-01-Single-Box-Random-7_bef2ee96-861e-41f5-9afb-b5650961e626_1200x.jpg?v=1631869901", //demon slayer
        */
    };
    char names[imageAmount][20] = {
        "image1.png",
        "image2.png",
        "image3.png",
        "image4.png",
        "image5.png",
        "image6.png",
        "image7.png",
        "image8.png",
        "image9.png",
        "image10.png"
    };

    // Get all the images from their URL
    CURL *curl_handle;
    char *imageFileName = (char*) malloc(sizeof(char) * 20);
    FILE *imageFile;
    curl_global_init(CURL_GLOBAL_ALL);
    curl_handle = curl_easy_init();
    //curl_easy_setopt(curl_handle, CURLOPT_VERBOSE, 1L);
    curl_easy_setopt(curl_handle, CURLOPT_NOPROGRESS, 1L);
    curl_easy_setopt(curl_handle, CURLOPT_WRITEFUNCTION, write_data);
    for (int i = 0; i < imageAmount; ++i) {
        printf("Looking for image %d...\n", i + 1);
        curl_easy_setopt(curl_handle, CURLOPT_URL, urls[i]);
        imageFile = fopen(names[i], "wb");
        if(imageFile) {
            curl_easy_setopt(curl_handle, CURLOPT_WRITEDATA, imageFile);
            curl_easy_perform(curl_handle);
            fclose(imageFile);
        }
        else {
            printf(" FAILED\n");
            return 1;
        }
        printf(" DONE\n");
    }
    printf("\n");
    curl_easy_cleanup(curl_handle);
    curl_global_cleanup();

    // Convolution kernel options
    float gaussianBlur[3][3] = {
        {0.0625*1, 0.0625*2, 0.0625*1},
        {0.0625*2, 0.0625*4, 0.0625*2},
        {0.0625*1, 0.0625*2, 0.0625*1}
    };
    float edgeDetection[3][3] = {
        {-1, -1, -1},
        {-1, 8, -1},
        {-1, -1, -1}
    };
    float example[3][3] = {
        {1, 0, -1},
        {1, 0, -1},
        {1, 0, -1}
    };

    // Execute all the operations for every image
    clock_t start;
    float execTime;
    float totalTime = 0;
    for(int f = 0; f < imageAmount; f++) {
        // Open image
        int width, height, componentCount;
        printf("Loading png file %d...\n", f + 1);
        unsigned char* inputData = stbi_load(names[f], &width, &height, &componentCount, 4);
        if (!inputData) {
            printf(" FAILED\n");
            return -1;
        }
        printf(" DONE\n" );

        // Convolution on CPU
        unsigned char* outputConvolution = (unsigned char*) malloc((width - 2) * (height - 2) * 4);
        printf("Applying convolution...\n");
        start = clock();
        convoluteCPU(inputData, outputConvolution, width, height, edgeDetection);
        execTime = ((float)(clock() - start)) / CLOCKS_PER_SEC;
        totalTime += execTime;
        printf(" DONE (");
        printTime(execTime);
        printf(")\n");

        // Pooling on CPU
        int poolWidth = (int)(width / POOLSTRIDE);
        int poolHeight = (int)(height / POOLSTRIDE);
        unsigned char* outputMaxPool = (unsigned char*) malloc(poolWidth * poolHeight * 4);
        unsigned char* outputMinPool = (unsigned char*) malloc(poolWidth * poolHeight * 4);
        unsigned char* outputAvgPool = (unsigned char*) malloc(poolWidth * poolHeight * 4);
        printf("Pooling...\n");
        start = clock();
        poolCPU(inputData, outputMaxPool, outputMinPool, outputAvgPool, width, height, POOLSTRIDE);
        execTime = ((float)(clock() - start)) / CLOCKS_PER_SEC;
        totalTime += execTime;
        printf(" DONE (");
        printTime(execTime);
        printf(")\n");

        // Write images back to disk
        printf("Writing output pngs to disk...\n");
        char buffer[255];
        memset(buffer, '\0', sizeof(buffer));
        strncpy(buffer, names[f], strlen(names[f]) - 4);
        strcat(buffer, "_convolution.png");
        stbi_write_png(buffer, width - 2, height - 2, 4, outputConvolution, 4 * width);
        memset(buffer, '\0', sizeof(buffer));
        strncpy(buffer, names[f], strlen(names[f]) - 4);
        strcat(buffer, "_maxpool.png");
        stbi_write_png(buffer, poolWidth, poolHeight, 4, outputMaxPool, 4 * poolWidth);
        memset(buffer, '\0', sizeof(buffer));
        strncpy(buffer, names[f], strlen(names[f]) - 4);
        strcat(buffer, "_minpool.png");
        stbi_write_png(buffer, poolWidth, poolHeight, 4, outputMinPool, 4 * poolWidth);
        memset(buffer, '\0', sizeof(buffer));
        strncpy(buffer, names[f], strlen(names[f]) - 4);
        strcat(buffer, "_avgpool.png");
        stbi_write_png(buffer, poolWidth, poolHeight, 4, outputAvgPool, 4 * poolWidth);
        printf(" DONE\n");

        // Free memory
        stbi_image_free(inputData);
        stbi_image_free(outputConvolution);
        stbi_image_free(outputMaxPool);
        stbi_image_free(outputMinPool);
        stbi_image_free(outputAvgPool);

        printf("\n");
    }
    printf("Total execution time of convolution and pooling on CPU: ");
    printTime(totalTime);
    printf("\n");

    printf("Total program execution time: ");
    printTime(((float)(clock() - programStart)) / CLOCKS_PER_SEC);
    printf("\n");
}
