#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define POOLSTRIDE 2

struct Pixel
{
    unsigned char r, g, b, a;
};

__global__ void convoluteGPU(unsigned char* input, unsigned char* output, int width, int height, float* kernel)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < height - 2 && col < width - 2) {
        int sum[4] = {0, 0, 0, 0};
        int opacity = 0;
        for (int i = 0; i < 3; ++i) {
            for (int j = 0; j < 3; ++j) {
                Pixel* p = (Pixel*)&input[((row + 1) + (i - 1)) * width * 4 + 4 * ((col + 1) + (j - 1))];
                sum[0] += p->r * kernel[i * 3 + j];
                sum[1] += p->g * kernel[i * 3 + j];
                sum[2] += p->b * kernel[i * 3 + j];
                sum[3] += p->a * kernel[i * 3 + j];
                if (i == 1 || j == 1)
                    opacity = p->a;
            }
        }
        Pixel* ptrPixel = (Pixel*)&output[row * width * 4 + 4 * col];
        for (int i = 0; i < 3; ++i) {
            if (sum[i] < 0)
                sum[i] = 0;
            else if (sum[i] > 255)
                sum[i] = 255;
        }
        ptrPixel->r = sum[0];
        ptrPixel->g = sum[1];
        ptrPixel->b = sum[2];
        ptrPixel->a = opacity;
    }
}

int main(int argc, char** argv)
{
    float gaussianBlur[3][3] = {
        {0.0625*1, 0.0625*2, 0.0625*1},
        {0.0625*2, 0.0625*4, 0.0625*2},
        {0.0625*1, 0.0625*2, 0.0625*1}
    };
    float edgeDetection[3][3] = {
        {-1, -1, -1},
        {-1, 8, -1},
        {-1, -1, -1}
    };
    float example[3][3] = {
        {1, 0, -1},
        {1, 0, -1},
        {1, 0, -1}
    };
    float *gaussianBlurGPU = nullptr;
    hipMalloc(&gaussianBlurGPU, 3 * 3 * sizeof(float));
    hipMemcpy(gaussianBlurGPU, gaussianBlur, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
    float *edgeDetectionGPU = nullptr;
    hipMalloc(&edgeDetectionGPU, 3 * 3 * sizeof(float));
    hipMemcpy(edgeDetectionGPU, edgeDetection, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
    float *exampleGPU = nullptr;
    hipMalloc(&exampleGPU, 3 * 3 * sizeof(float));
    hipMemcpy(exampleGPU, example, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

    // Open image
    int width, height, componentCount;
    printf("Loading png file...\r\n");
    unsigned char* inputData = stbi_load(argv[1], &width, &height, &componentCount, 4);
    if (!inputData)
    {
        printf("Failed to open image\r\n");
        return -1;
    }
    printf(" DONE \r\n" );

    // Copy data to the GPU
    printf("Copy data to GPU...\r\n");
    unsigned char* inputDataGPU = nullptr;
    hipMalloc(&inputDataGPU, width * height * 4);
    hipMemcpy(inputDataGPU, inputData, width * height * 4, hipMemcpyHostToDevice);
    printf(" DONE \r\n");

    // Process image on GPU
    unsigned char* outputConvolutionGPU = nullptr;
    hipMalloc(&outputConvolutionGPU, (width - 2) * (height - 2) * 4);
    printf("Running CUDA Kernel...\r\n");
    dim3 blockSize(32, 32);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    convoluteGPU<<<gridSize, blockSize>>>(inputDataGPU, outputConvolutionGPU, width, height, edgeDetectionGPU);
    hipDeviceSynchronize();
    printf(" DONE \r\n" );

    // Copy data from the GPU
    printf("Copy data from GPU...\r\n");
    unsigned char* outputConvolution = (unsigned char*) malloc(sizeof(unsigned char) * (width - 2) * (height - 2) * 4);
    hipMemcpy(outputConvolution, outputConvolutionGPU, (width - 2) * (height - 2) * 4, hipMemcpyDeviceToHost);
    printf(" DONE \r\n");

    // Write images back to disk
    printf("Writing pngs to disk...\r\n");
    stbi_write_png("convolutionGPU.png", width - 2, height - 2, 4, outputConvolution, 4 * width);
    printf(" DONE\r\n");

    // Free memory
    hipFree(inputDataGPU);
    hipFree(outputConvolutionGPU);
    stbi_image_free(inputData);
    stbi_image_free(outputConvolution);

    hipFree(gaussianBlurGPU);
    hipFree(edgeDetectionGPU);
    hipFree(exampleGPU);
}
