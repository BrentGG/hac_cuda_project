#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

struct Pixel
{
 unsigned char r, g, b, a;
};

void ConvertImageToGrayCpu(unsigned char* imageRGBA, int width, int height)
{
  long int avgPixelR=0;
  int maxPixelR=0;
  int minPixelR=0;

  long int avgPixelG=0;
  int maxPixelG=0;
  int minPixelG=0;

  long int avgPixelB=0;
  int maxPixelB=0;
  int minPixelB=0;

    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            Pixel* ptrPixel = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
            unsigned char pixelValue = (unsigned char)(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);


            //wip ideals deel 2 min/max/avg pooling

            //avg pooling RED
            avgPixelR=avgPixelR+ptrPixel->r;
              avgPixelG=avgPixelG+ptrPixel->g;
                avgPixelB=avgPixelB+ptrPixel->b;
            //max pooling RED
            if (ptrPixel->r > maxPixelR)
            {
              maxPixelR=ptrPixel->r;
            }
            //min pooling RED
            if (ptrPixel->r < minPixelR)
            {
              minPixelR=ptrPixel->r;
            }
            //max pooling GREEN
            if (ptrPixel->g > maxPixelG)
            {
              maxPixelG=ptrPixel->g;
            }
            //min pooling GREEN
            if (ptrPixel->g < minPixelG)
            {
              minPixelG=ptrPixel->g;
            }
              //max pooling BLUE
            if (ptrPixel->b > maxPixelB)
            {
              maxPixelB=ptrPixel->b;
            }
            //min pooling BLUE
            if (ptrPixel->b < minPixelB)
            {
              minPixelB=ptrPixel->b;
            }
        }
    }
    avgPixelR= avgPixelR/(height*width);
    avgPixelG= avgPixelG/(height*width);
    avgPixelB= avgPixelB/(height*width);
    printf("avg pooling for red: %li ,green: %li and blue: %li\n", avgPixelR ,avgPixelG ,avgPixelB );
    printf("max pooling for red: %d ,green: %d and blue: %d\n", maxPixelR, maxPixelG, maxPixelB );
    printf("min pooling for red: %d ,green: %d and blue: %d\n", minPixelR, minPixelG, minPixelB );

}


__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   int idy = blockIdx.y * blockDim.y + threadIdx.y;

   Pixel* ptrPixel = (Pixel*)&imageRGBA[idy * gridDim.x*blockDim.x * 4 + 4 * idx];
            unsigned char pixelValue = (unsigned char)(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
            ptrPixel->r = pixelValue;
            ptrPixel->g = pixelValue;
            ptrPixel->b = pixelValue;
            ptrPixel->a = 255;


  // TODO
}

int main(int argc, char** argv)
{
    // Check argument count
    if (argc < 2)
    {
        printf("Usage: im2gray <filename>\r\n");
        return -1;
    }

    // Open image
    int width, height, componentCount;
    printf("Loading png file...\r\n");
    unsigned char* imageData = stbi_load(argv[1], &width, &height, &componentCount, 4);
    if (!imageData)
    {
        printf("Failed to open Image\r\n");
        return -1;
    }
    printf(" DONE \r\n" );


    // Validate image sizes
    if (width % 32 || height % 32)
    {
        // NOTE: Leaked memory of "imageData"
        printf("Width and/or Height is not dividable by 32!\r\n");
        return -1;
    }


    // Process image on cpu
    printf("Processing image...\r\n");
    ConvertImageToGrayCpu(imageData, width, height);
    printf(" DONE \r\n");

    // Copy data to the gpu
    printf("Copy data to GPU...\r\n");
    unsigned char* ptrImageDataGpu = nullptr;
    hipMalloc(&ptrImageDataGpu, width * height * 4);
    hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice);
    printf(" DONE \r\n");

    // Process image on gpu
    printf("Running CUDA Kernel...\r\n");
    dim3 blockSize(32, 32);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    //ConvertImageToGrayGpu<<<gridSize, blockSize>>>(ptrImageDataGpu);
    printf(" DONE \r\n" );

    // Copy data from the gpu
    printf("Copy data from GPU...\r\n");
    hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost);
    printf(" DONE \r\n");

    // Build output filename
    const char * fileNameOut = "gray.png";

    // Write image back to disk
    printf("Writing png to disk...\r\n");
    stbi_write_png(fileNameOut, width, height, 4, imageData, 4 * width);
    printf("DONE\r\n");

    // Free memory
    hipFree(ptrImageDataGpu);
    stbi_image_free(imageData);
}
